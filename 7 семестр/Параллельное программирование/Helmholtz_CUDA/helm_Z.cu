
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>


#define blocksizeX 512// blocksizeX*blocksizeY == (n-2)/2 т.к. не считаем в гу
#define blocksizeY 1
#define n 1026 //количество узлов
#define iter_end 100000//кол-во итераций
#define k 100.0
#define pi 3.1415926535897932385


using namespace std;

typedef double mytipe;  //тип данных, использующийся во всей программе

__constant__ mytipe c[2];//коэффиценты в расчётной формуле запишем в константную память


mytipe f(mytipe x, mytipe y)
{
	return 2.*sin(pi*y) + k*k*(1. - x)*x*sin(pi*y) + pi*pi*(1. - x)*x*sin(pi*y);
}


__global__ void KernelBlack(mytipe* y_red, mytipe*  y_black,mytipe* f_black)
{

	// глобальное положение элемента в матрице
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;//строка и столбец элемента текущей нити
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;//пересчитываются только внутр эл-ты (без ГУ)

	int i  = row * n/2 + col;//текущий индекс и индексы соседних элементов
	int up = (row + 1) * n/2 + col;
	int down  = (row - 1) * n/2 + col;
	int left = row * n/2 + col;
	int right = row * n/2 + col + 1;


	//каждый поток вычисляет один раз
        if ((row % 2)==0){
		y_black[i - 1] = f_black[i - 1]*c[0] + (y_red[up - 1] + y_red[down - 1] + y_red[left - 1] + y_red[right - 1])*c[1];
	} else {
        	y_black[i] = f_black[i]*c[0] + (y_red[up] + y_red[down] + y_red[left - 1] + y_red[right - 1])*c[1];
        };

       /* if ((row % 2)==0){
		y_black[i - 1] = f(row*h,(2*col - 1)*h);
	} else {
        	y_black[i] = f(row*h,2*col*h);
        };*/
	
	return;
}

__global__ void KernelRed(mytipe* y_red, mytipe*  y_black,mytipe* f_red)
{

	// глобальное положение элемента в матрице
	int col = blockIdx.x * blockDim.x + threadIdx.x + 1;//строка и столбец элемента текущей нити
	int row = blockIdx.y * blockDim.y + threadIdx.y + 1;//пересчитываются только внутр эл-ты (без ГУ)

	int i  = row * n/2 + col;//текущий индекс и индексы соседних элементов
	int up = (row + 1) * n/2 + col;
	int down  = (row - 1) * n/2 + col;
	int left = row * n/2 + col;
	int right = row * n/2 + col + 1;

	//каждый поток вычисляет один раз
         if ((row % 2)==0){
		y_red[i] = f_red[i]*c[0] + (y_black[up] + y_black[down] + y_black[left-1] + y_black[right-1])*c[1];
        } else {
        	y_red[i - 1] = f_red[i - 1]*c[0] + (y_black[up - 1] + y_black[down - 1] + y_black[left - 1] + y_black[right - 1])*c[1];
        };

/*if ((row % 2)==0){
		y_red[i] = f(row*h,2*col*h);
	} else {
        	y_red[i - 1]= f(row*h,(2*col - 1)*h);
        };*/

		
	return;
}


mytipe u(mytipe x, mytipe y)
{
	return (1. - x)*x*sin(pi*y);
}

mytipe error (mytipe* y_red, mytipe* y_black, mytipe h)
{
	mytipe err = 0.0;
	mytipe maxerr_r = 0.0;
	mytipe maxerr_b = 0.0;
	int m;
	
	for (int i = 0; i < n; i++){
		m =0;
	    for (int j = (i % 2); j < n; j += 2) {
			
			err = fabs(y_red[i*n/2 + m] - u(i*h, j*h));
			if (err > maxerr_r)
				maxerr_r = err;
		++m; 
		}
	}
	
	
	for (int i = 0; i < n; i++){ 
		m=0;
		for (int j = ((i + 1) % 2); j < n; j += 2) {
			err = fabs(y_black[i*n/2 + m] - u(i*h, j*h));
			if (err > maxerr_b)
				maxerr_b = err; 
		++m;
		}
	}
	
	if(maxerr_r > maxerr_b) return maxerr_r;
	else return maxerr_b;
	

}


int main() {
	
	mytipe h = 1. / mytipe(n - 1);//делим отрезок на кол-во разбиений

	mytipe* y_red = new mytipe [n*n/2];
	mytipe* y_black = new mytipe [n*n/2];
	mytipe* F_red = new mytipe [n*n/2];
	mytipe* F_black = new mytipe [n*n/2];
			
	
	for(int i = 0; i < n; i++) 
		for(int j = 0; j < n/2; j++) {y_black[i*n/2+j]=0.;y_red[i*n/2+j]=0.;}
	
	
	int m;
	
	for (int i = 0; i < n; i++){
		m =0;
	        for (int j = (i % 2); j < n; j += 2) { F_red[i*n/2+m] =f(i*h,j*h); ++m;}
	}
     
	for (int i = 0; i < n; i++){ 
		 m=0;
		for (int j = ((i + 1) % 2); j < n; j += 2) { F_black[i*n/2+m] =f(i*h,j*h); ++m;}
	}			
	
	
	
	hipError_t SD;

	SD = hipSetDevice(0);
	if (SD != hipSuccess)//проверяем подключилась ли графическая карта
	{
		cout << "CUDA set device error" << endl;
		return 1;
	}
	
	//создаем указатели на графическом ядре (девайсе)
        mytipe* dev_y_red = NULL;
	mytipe* dev_y_black = NULL;
	mytipe* dev_F_red = NULL;
	mytipe* dev_F_black = NULL;
	
	int nbytes = n*n*sizeof(mytipe)/2;

	//выделяем под них память
	hipMalloc ((void **)&dev_y_red, nbytes);
	hipMalloc ((void **)&dev_y_black, nbytes);
	hipMalloc ((void **)&dev_F_red, nbytes);
        hipMalloc ((void **)&dev_F_black, nbytes);
	
	dim3 threads(blocksizeX, blocksizeY);//кол-во тредов под один блок
	dim3 blocks((n-2)/blocksizeX/2, (n-2)/blocksizeY);//кол-во блоков. (n-2) т.к. на границе ничего не вычисляем
	
	hipEvent_t start, stop;//счетчики время
	hipEventCreate(&start);
	hipEventCreate(&stop);
		
	hipEventRecord(start,0);
	hipEventSynchronize(start);
	
	hipMemcpy(dev_y_red, y_red, nbytes, hipMemcpyHostToDevice);//копирование данных  с хоста на девайс 
	hipMemcpy(dev_y_black, y_black, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dev_F_red, F_red, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dev_F_black, F_black, nbytes, hipMemcpyHostToDevice);
	mytipe host_c[2] =  {h*h / (4.0 + h*h*k*k),1.0 / (4.0 + h*h*k*k)};
	hipMemcpyToSymbol(HIP_SYMBOL(c),host_c,2*sizeof(mytipe),0, hipMemcpyHostToDevice);//передача константной памяти

	
	for(int iter = 0; iter < iter_end; iter++)
	{            	
             KernelBlack<<<blocks,threads>>>(dev_y_red,dev_y_black,dev_F_black);
            // cudaDeviceSynchronize();
             KernelRed<<<blocks,threads>>>(dev_y_red,dev_y_black,dev_F_red);
	    // cudaDeviceSynchronize();
        }



	hipMemcpy(y_red, dev_y_red, nbytes, hipMemcpyDeviceToHost);
	hipMemcpy(y_black, dev_y_black, nbytes,hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);	
	hipEventSynchronize(stop);



	float dt;
	hipEventElapsedTime(&dt,start,stop);
	cout << "dim "<< n << "x" << n<< endl;
	cout << "iter "<< iter_end<< endl;
	cout << "error  "<< error(y_red, y_black, h)<<endl;
	cout << "time " << dt/1000 << " seconds"<< endl;	
	cout<<"blocksize x: "<<blocksizeX<<" y: "<<blocksizeY<<endl;

	
	delete[] y_red;
	delete[] y_black;
	delete[] F_red;
	delete[] F_black;

	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_y_red);
	hipFree(dev_y_black);
	hipFree(dev_F_red);
	hipFree(dev_F_black);
	
	return 0;

}