
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>


#define blocksizeX 32// blocksizeX*blocksizeY == n-2 т.к. не считаем в гу
#define blocksizeY 32
#define n 1026 //количество узлов
#define iter_end 100000//кол-во итераций
#define k 100.0
#define pi 3.1415926535897932385


using namespace std;

typedef double mytipe;  //тип данных, использующийся во всей программе

__constant__ mytipe c[2];//коэффиценты в расчётной формуле запишем в константную память


__global__ void KernelJacobe(mytipe* yk, mytipe*  yk1,mytipe* f)
{
	int row,col;// глобальное положение элемента в матрице
	col = blockIdx.x * blockDim.x + threadIdx.x + 1;//строка и столбец элемента текущей нити
	row = blockIdx.y * blockDim.y + threadIdx.y + 1;//пересчитываются только внутр эл-ты (без ГУ)

	int i  = row * n + col;//текущий индекс и индексы соседних элементов
	int up = (row + 1) * n + col;
	int down  = (row - 1) * n + col;
	int left = row * n + col - 1;
	int right = row * n + col + 1;

	//каждый поток вычисляет один раз
	yk1[i]=f[i]*c[0]+(yk[up]+yk[down]+yk[left]+yk[right])*c[1];
		
	return;
}


mytipe f(mytipe x, mytipe y)
{
	return 2.*sin(pi*y) + k*k*(1. - x)*x*sin(pi*y) + pi*pi*(1. - x)*x*sin(pi*y);
}

mytipe u(mytipe x, mytipe y)
{
	return (1. - x)*x*sin(pi*y);
}

mytipe error (mytipe* yk1, mytipe h)
{
	mytipe err = 0.0;
	mytipe maxerr = 0.0;
	
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
		{
			err = fabs(yk1[i*n + j] - u(i*h, j*h));
			if (err > maxerr)
				maxerr = err;
		}
	
	return maxerr;

}


int main() {
	
	mytipe h = 1. / mytipe(n - 1);//делим отрезок на кол-во разбиений

	mytipe* yk = new mytipe [n*n];
	mytipe* yk1 = new mytipe [n*n];
	mytipe* F = new mytipe [n*n];
	
	for(int i = 0; i < n; i++) 
		for(int j = 0; j < n; j++) {yk[i*n+j]=0.;yk1[i*n+j]=0.;F[i*n+j]=f(i*h,j*h);}

	int nbytes = n*n*sizeof(mytipe);
	
	
	hipError_t SD;

	SD = hipSetDevice(0);
	if (SD != hipSuccess)//проверяем подключилась ли графическая карта
	{
		cout << "CUDA set device error" << endl;
		return 1;
	}
	
	//создаем указатели на графическом ядре (девайсе)
    mytipe* dev_yk = NULL;
	mytipe* dev_yk1 = NULL;
	mytipe* dev_F = NULL;

	//выделяем под них память
	hipMalloc ((void **)&dev_yk, nbytes);
	hipMalloc ((void **)&dev_yk1, nbytes);
	hipMalloc ((void **)&dev_F, nbytes);

	
	dim3 threads(blocksizeX, blocksizeY);//кол-во тредов под один блок
	dim3 blocks((n-2)/blocksizeX, (n-2)/blocksizeY);//кол-во тредов под один блок. (n-2) т.к. на границе ничего не вычисляем
	
	hipEvent_t start, stop;//счетчики время
	hipEventCreate(&start);
	hipEventCreate(&stop);
		
	hipEventRecord(start,0);
	hipEventSynchronize(start);
	
	hipMemcpy(dev_yk, yk, nbytes, hipMemcpyHostToDevice);//копирование данных  с хоста на девайс 
	hipMemcpy(dev_yk1, yk1, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dev_F, F, nbytes, hipMemcpyHostToDevice);
	mytipe host_c[2] =  {h*h / (4.0 + h*h*k*k),1.0 / (4.0 + h*h*k*k)};
	hipMemcpyToSymbol(HIP_SYMBOL(c),host_c,2*sizeof(mytipe),0, hipMemcpyHostToDevice);//передача константной памяти

	
	for(int iter = 0; iter < iter_end; iter++)
	{
		KernelJacobe<<<blocks,threads>>>(dev_yk,dev_yk1,dev_F);
		swap(dev_yk, dev_yk1);
	}



	hipMemcpy(yk1,dev_yk1,nbytes,hipMemcpyDeviceToHost);
	//cudaMemcpy(yk,dev_yk,nbytes,cudaMemcpyDeviceToHost);
	hipEventRecord(stop,0);	
	hipEventSynchronize(stop);
	
	float dt;
	hipEventElapsedTime(&dt,start,stop);
	cout << "dim "<< n << "x" << n<< endl;
	cout << "iter "<< iter_end<< endl;
	cout << "error  "<< error(yk1,h)<<endl;
	cout << "time " << dt/1000 << " seconds"<< endl;	
	cout<<"blocksize x: "<<blocksizeX<<" y: "<<blocksizeY<<endl;

	
	delete[] yk;
	delete[] yk1;
	delete[] F;

	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_yk);
	hipFree(dev_yk1);
	hipFree(dev_F);
	
	return 0;

}



/*__global__ void KernelJacobeLoc(double* yk, double*  yk1,double* f)
{
	int row,col;// глобальное положение элемента в матрице
	col = blockIdx.x * blockDim.x + threadIdx.x + 1;//строка и столбец элемента текущей нити
	row = blockIdx.y * blockDim.y + threadIdx.y + 1;//пересчитываются только внутр эл-ты (без ГУ)

	int i  = row * n + col;//текущий индекс и индексы соседних элементов
	int up = (row + 1) * n + col;
	int down  = (row - 1) * n + col;
	int left = row * n + col - 1;
	int right = row * n + col + 1;
	
	const int Nx_loc = blocksizeX+2;//(+2 для ГУ)
	const int Ny_loc = blocksizeY+2;

	__shared__ double  yk_loc[Nx_loc*Ny_loc];//лок матрица под блок 
	
	int col_loc = threadIdx.x + 1;//+1 т.к. в гу не считаем
	int row_loc = threadIdx.x + 1;
	
	//int i_loc  = row_loc * Nx_loc + col_loc;//текущий индекс и индексы соседних элементов в лок матр
	int up_loc = (row_loc + 1)* Nx_loc + col_loc;
	int down_loc  = (row_loc - 1) * Nx_loc + col_loc;
	int left_loc = row_loc * Nx_loc + col_loc - 1;
	int right_loc = row_loc * Nx_loc + col_loc + 1;
	
	yk_loc[up_loc] = yk[up];
	yk_loc[down_loc] = yk[down];
	yk_loc[left_loc] = yk[left];
	yk_loc[right_loc] = yk[right];
	
	//каждый поток вычисляет один раз
	yk1[i] = f[i]*c[0] + (yk_loc[up_loc] + yk_loc[down_loc] + yk_loc[left_loc] + yk_loc[right_loc])*c[1];
		
	return;
}*/